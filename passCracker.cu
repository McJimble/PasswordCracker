#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

//#define __DEBUG

#define NUM_TESTS 1
#define N 4096*4096
#define THREADS_PER_BLOCK 128
#define PASS_LENGTH 5
#define ALPHABET_SIZE 81
//       ^
//       |  Make sure these match in size if changing either!
//       v
// This alphabet is based on IBM's valid password characters
const char* possibleChars = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789~`!@#$%^&*()_-+=:;?";
const int totalBlocks = N / THREADS_PER_BLOCK;

// These never change and are pre-computed before kernel execution.
// Could have this data be #define constants, but I want this to be adaptable
// to any alphabet, password size, and thread count.
__constant__ int c_GuessesPerThread;
__constant__ long c_MaxGuesses;
__constant__ char c_Alphabet[ALPHABET_SIZE];
__constant__ char c_Password[PASS_LENGTH];

// A fast device version of string comparison. For efficiency sake, however, just
// returns 0 for non-equal, and 1 for equal. Less checks this way.
__device__ int GPU_StrCmp(const char* str1, const char* str2, unsigned size)
{
    for (unsigned i = 0; i < size; i++)
        if(str1[i] != str2[i]) return 0;

    return 1;
}

// Makes each thread compute a certain number of guesses. If any one thread's guess
// matches, throw a flag that makes all other threads stop.
__global__ void GPU_CrackKernel(int* stopFlag, long* totalGuesses)
{
#ifdef __DEBUG
    __shared__ long guessesThisBlock[THREADS_PER_BLOCK];
#endif
    // Putting data into registers since they are referred to multiple times.
    // should be faster?
    int guessesPerThread = c_GuessesPerThread;
    long maxGuesses = c_MaxGuesses;

    // Starting guess number out of maximum guess count
    long guess = (long)guessesPerThread * (threadIdx.x + (blockIdx.x * blockDim.x));

    // Same algorithm as CPU, but per-thread. Each thread will only computer a certain
    // number of guesses in parallel. 
    char guessStr[PASS_LENGTH];
    int guessesThisThread = 0;
    while(guessesThisThread < guessesPerThread && guess < maxGuesses)
    {
        long temp = guess;

        // Solving 5+ character passwords is trivial, so we
        // unroll the first 5 characters of the loop.
        guessStr[0] = c_Alphabet[temp % ALPHABET_SIZE];
        temp /= ALPHABET_SIZE;

        guessStr[1] = c_Alphabet[temp % ALPHABET_SIZE];
        temp /= ALPHABET_SIZE;

        guessStr[2] = c_Alphabet[temp % ALPHABET_SIZE];
        temp /= ALPHABET_SIZE;

        guessStr[3] = c_Alphabet[temp % ALPHABET_SIZE];
        temp /= ALPHABET_SIZE;

        guessStr[4] = c_Alphabet[temp % ALPHABET_SIZE];
        temp /= ALPHABET_SIZE;
        
        for (int i = 5; i < PASS_LENGTH; i++)
        {
            guessStr[i] = c_Alphabet[temp % ALPHABET_SIZE];
            temp /= ALPHABET_SIZE;
        }

        if (GPU_StrCmp(guessStr, c_Password, PASS_LENGTH) == 1) *stopFlag = 1;
        if (*stopFlag == 1) break;

        guess++;
        guessesThisThread++;
    }
#ifdef __DEBUG
    guessesThisBlock[threadIdx.x] = guessesThisThread;

    // Add up total number of guesses per block.
    __syncthreads();
    if (threadIdx.x == 0)
	{
        long sum = 0;
		for (int i = 0; i < THREADS_PER_BLOCK; i++)
		{
			sum += guessesThisBlock[i];
		}
		totalGuesses[blockIdx.x] = sum;
	}
#endif
}

void CreateRandomPassword(int length, char* pass)
{
    srand(time(0));
    for(int i = 0; i < length; i++)
        pass[i] = possibleChars[rand() % ALPHABET_SIZE];

    printf("\nPassword: %s\n", pass);
}

long long TimevalToMilliseconds(struct timeval tv)
{
    return tv.tv_sec * 1000000 + tv.tv_usec;
}

// Sequentially tries every possible combination of alpha-numeric characters
// and symbols until the alloted time halted or it solves the password.
// We assume password will NOT be solved. We just want to know how
// many guesses it takes the CPU in the same time the GPU took to solve it!
int CPU_CrackPassword(const char* password)
{
    struct timeval end;
    struct timeval start;
    gettimeofday(&start, NULL);

    printf("\nStarting CPU crack...\n");

    char guessStr[PASS_LENGTH];
    long totalGuesses = 1;
    for (int i = 0; i < PASS_LENGTH; i++)
    {
        totalGuesses *= ALPHABET_SIZE;
        guessStr[i] = possibleChars[0];
    }

    long guess;
    for (guess = 0; guess < totalGuesses; guess++)
    {
        // Chars go to next if the previous one flipped from the last to first
        // element. Number of guesses required to set character to next increases
        // in order of magnitude (alphabetSize)^n for each character n.
        long temp = guess;
        for (int i = 0; i < PASS_LENGTH; i++)
        {
            guessStr[i] = possibleChars[temp % ALPHABET_SIZE];
            temp /= ALPHABET_SIZE;
        }

        //printf("Guess: %ld Password: %s\n", guess, guessStr);

        if (memcmp(guessStr, password, PASS_LENGTH) == 0)
        {
            printf("Password found on CPU!\n");
            break;
        }
    }

    gettimeofday(&end, NULL);
    float milliseconds = (TimevalToMilliseconds(end) - TimevalToMilliseconds(start)) / 1000;

    printf("CPU computed %ld guesses in %.5f seconds\n", guess, milliseconds / 1000);
    return milliseconds;
}

float GPU_CrackPassword(const char* pass)
{
    printf("\nStarting GPU crack...\n");

    // Host vars.
    long *totalGuesses;
    long *maxGuesses;
    int *guessesPerThread;
    char *alphabet;
    char *password;
    
    long *d_totalGuesses;
    int *d_foundFlag;

    hipEvent_t start, stop;    // Cuda flags for timing.
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Global vars & relevant host vars.
    hipMalloc(((void**)&d_foundFlag), sizeof(int));
    hipMalloc(((void**)&d_totalGuesses), sizeof(long) * totalBlocks);
    
    totalGuesses = (long*)malloc(sizeof(long) * totalBlocks);
    maxGuesses = (long*)malloc(sizeof(long));
    guessesPerThread = (int*)malloc(sizeof(int));
    alphabet = (char*)malloc(sizeof(char) * ALPHABET_SIZE);
    password = (char*)malloc(sizeof(char) * PASS_LENGTH);

    *maxGuesses = (long)pow(ALPHABET_SIZE, PASS_LENGTH);
    *guessesPerThread = (int)ceil((double)*maxGuesses / (N));

    for(int i = 0; i < ALPHABET_SIZE; i++)
        alphabet[i] = possibleChars[i];
    for(int i = 0; i < PASS_LENGTH; i++)
        password[i] = pass[i];

    hipMemcpyToSymbol(HIP_SYMBOL(c_MaxGuesses), maxGuesses, sizeof(long));
    hipMemcpyToSymbol(HIP_SYMBOL(c_GuessesPerThread), guessesPerThread, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_Alphabet), alphabet, sizeof(char) * ALPHABET_SIZE);
    hipMemcpyToSymbol(HIP_SYMBOL(c_Password), password, sizeof(char) * PASS_LENGTH);

    printf("Maximum guesses to compute: %ld\nGuesses per GPU thread: %d\n", *maxGuesses, *guessesPerThread);

    // Call kernel + time with events.
    hipEventRecord(start);

    GPU_CrackKernel<<< totalBlocks, THREADS_PER_BLOCK >>>(d_foundFlag, d_totalGuesses);

    hipEventRecord(stop);

    // Get total guesses, calculate time it took to solve.
    hipMemcpy(totalGuesses, d_totalGuesses, sizeof(long) * totalBlocks, hipMemcpyDeviceToHost);
    long sum = 0;
    for (int i = 0; i < totalBlocks; i++)
        sum += totalGuesses[i];

    if (sum > 0)
        printf("Total Guesses made: %ld\n", sum);
    printf("\n");

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU found password in: %.5f sec\n", (milliseconds / 1000));

    free(totalGuesses);
    free(maxGuesses);
    free(guessesPerThread);
    free(alphabet);
    free(password);

    hipFree(d_totalGuesses);
    hipFree(d_foundFlag);

    return milliseconds;
}

int main(int argc, char *argv[])
{
    // Check for running CPU flag.
    int runCPU = 0;
    if (argc == 2)
        if (!strcmp(argv[1], "-cpu")) runCPU = 1;

#ifdef __DEBUG
    FILE *fp;
    fp = fopen("Results.txt", "w+");
#endif
    char* password = (char*)malloc(sizeof(char) * PASS_LENGTH);

    // Call CPU/GPU functions specified number of times with random password.
    for (int i = 0; i < NUM_TESTS; i++)
    {
        CreateRandomPassword(PASS_LENGTH, password);

        float gpuBenchmark = GPU_CrackPassword(password);
        if (runCPU == 1)
            float cpuBenchmark = CPU_CrackPassword(password);
        
        printf("\n******\n");

#ifdef __DEBUG
        fprintf(fp, "%.5f\n", gpuBenchmark / 1000);
#endif
    }

    free(password);
    return -1;
}